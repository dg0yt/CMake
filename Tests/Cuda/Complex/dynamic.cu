
#include <string>
#include <hip/hip_runtime.h>
#include <iostream>

#ifdef _WIN32
#define EXPORT __declspec(dllexport)
#else
#define EXPORT
#endif

int dynamic_base_func(int);

EXPORT int __host__ cuda_dynamic_host_func(int x)
{
  return dynamic_base_func(x);
}

static
__global__
void DetermineIfValidCudaDevice()
{
}

EXPORT void cuda_dynamic_lib_func()
{
  DetermineIfValidCudaDevice <<<1,1>>> ();
  hipError_t err = hipGetLastError();
  if(err != hipSuccess)
    {
    std::cerr << "DetermineIfValidCudaDevice [SYNC] failed: "
              << hipGetErrorString(err) << std::endl;
    }
  err = hipDeviceSynchronize();
  if(err != hipSuccess)
    {
    std::cerr << "DetermineIfValidCudaDevice [ASYNC] failed: "
              << hipGetErrorString(hipGetLastError()) << std::endl;
    }
}
